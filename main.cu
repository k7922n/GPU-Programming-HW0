#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

__global__ void SomeTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < fsize && input_gpu[idx] != '\n') {
		input_gpu[idx] = '!';
	}
}

__global__ void MyTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < fsize && input_gpu[idx] != '\n') {
		if(int(input_gpu[idx]) > 64 && input_gpu[idx] < 91)
			{input_gpu[idx] = char(int(input_gpu[idx]) + 32);}
		else if(int(input_gpu[idx]) > 96 && input_gpu[idx] < 123)
			{input_gpu[idx] = char(int(input_gpu[idx]) - 32);}
	}
}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (! fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
	//SomeTransform<<<100, 64>>>(input_gpu, fsize);
	
	//MyTransform turns the lower case into upper case and turns the upper case into lower one
	MyTransform<<<100, 64>>>(input_gpu, fsize);
	puts(text_smem.get_cpu_ro());
	return 0;
}
